
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int gridStride = gridDim.x * blockDim.x;
  
  for(i; i < N; i+=gridStride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  checkCuda(hipMallocManaged(&a,size));
  checkCuda(hipMallocManaged(&b,size));
  checkCuda(hipMallocManaged(&c,size));

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);
  
  const int threads_per_block = 32;
  const int num_blocks = 256;
  
  clock_t begin = clock();
  addVectorsInto<<<num_blocks, threads_per_block>>>(c, a, b, N);
  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  printf("Elapsed time for addVectorsInto on GPU is %lf \n", time_spent);
  
  checkCuda(hipDeviceSynchronize());
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
